#include "gemm.cuh"

int main(const int argc, char *argv[]) {
    if (argc != 2) {
        std::cout << "Usage: " << argv[0] << " <N>" << std::endl;
    }

    const auto N = static_cast<int>(strtol(argv[1], nullptr, 10));
    std::cout << "Matrix size: " << N << std::endl;

    const int m = N;
    const int n = N;
    const int k = N;
    const int lda = N;
    const int ldb = N;
    const int ldc = N;
    constexpr hipblasOperation_t transA = HIPBLAS_OP_N;
    constexpr hipblasOperation_t transB = HIPBLAS_OP_N;
    constexpr double alpha = 1.0;
    constexpr double beta = 0.0;
    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;
    std::vector<data_type> A(N * N);
    std::vector<data_type> B(N * N);
    std::vector<data_type> C(N * N);
    data_type *d_A = nullptr;
    data_type *d_B = nullptr;
    data_type *d_C = nullptr;

    matrixInitialization(A.data(), B.data(), N);

    // Initialize CUBLAS handle and bind the stream
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUDA_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CALL(hipblasSetStream(cublasH, stream));

    CUDA_CALL(hipMalloc(&d_A, A.size() * sizeof(data_type)));
    CUDA_CALL(hipMalloc(&d_B, B.size() * sizeof(data_type)));
    CUDA_CALL(hipMalloc(&d_C, C.size() * sizeof(data_type)));

    START_CUDA_TIMER(Gemm)
        CUDA_CALL(hipMemcpyAsync(d_A, A.data(), A.size() * sizeof(data_type), hipMemcpyHostToDevice, stream));
        CUDA_CALL(hipMemcpyAsync(d_B, B.data(), B.size() * sizeof(data_type), hipMemcpyHostToDevice, stream));

        CUBLAS_CALL(hipblasDgemm(cublasH, transA, transB, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc));

        CUDA_CALL(hipMemcpyAsync(C.data(), d_C, C.size() * sizeof(data_type), hipMemcpyDeviceToHost, stream));
        CUDA_CALL(hipStreamSynchronize(stream));
    STOP_CUDA_TIMER_PRINT_ELAPSED(Gemm)

    CUDA_CALL(hipFree(d_A));
    CUDA_CALL(hipFree(d_B));
    CUDA_CALL(hipFree(d_C));

    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUDA_CALL(hipStreamDestroy(stream));

    CUDA_CALL(hipDeviceReset());

    return EXIT_SUCCESS;
}

void matrixInitialization(double *a, double *b, const int n) {
    assert_non_null(a);
    assert_non_null(b);

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            a[i * n + j] = randomInRange(1, 1000);
            b[i * n + j] = randomInRange(1, 1000);
        }
    }
}
