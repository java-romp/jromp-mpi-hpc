#include "big_multiplication.cuh"

int main(const int argc, char *argv[]) {
    if (argc != 2) {
        std::cout << "Usage: " << argv[0] << " <N>" << std::endl;
    }

    const auto N = static_cast<size_t>(strtol(argv[1], nullptr, 10));
    std::cout << "Matrix size: " << N << std::endl;

    double *d_A, *d_B, *d_C;
    constexpr double alpha = 1.0;
    constexpr double beta = 0.0;
    hipblasHandle_t cublasH = nullptr;
    hipStream_t stream = nullptr;

    auto *A = static_cast<double *>(malloc(N * N * sizeof(double)));
    auto *B = static_cast<double *>(malloc(N * N * sizeof(double)));
    auto *C = static_cast<double *>(malloc(N * N * sizeof(double)));

    matrixInitialization(A, B, N);

    // Initialize CUBLAS handle and bind the stream
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUDA_CALL(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUBLAS_CALL(hipblasSetStream(cublasH, stream));

    CUDA_CALL(hipMalloc(&d_A, N * N * sizeof(double)));
    CUDA_CALL(hipMalloc(&d_B, N * N * sizeof(double)));
    CUDA_CALL(hipMalloc(&d_C, N * N * sizeof(double)));

    START_CUDA_TIMER(Gemm)
        CUDA_CALL(hipMemcpyAsync(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice, stream));
        CUDA_CALL(hipMemcpyAsync(d_B, B, N * N * sizeof(double), hipMemcpyHostToDevice, stream));

        CUBLAS_CALL(hipblasDgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N));

        CUDA_CALL(hipMemcpyAsync(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost, stream));
        CUDA_CALL(hipStreamSynchronize(stream));
    STOP_CUDA_TIMER_PRINT_ELAPSED(Gemm)

    free(A);
    free(B);
    free(C);
    CUDA_CALL(hipFree(d_A));
    CUDA_CALL(hipFree(d_B));
    CUDA_CALL(hipFree(d_C));

    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUDA_CALL(hipStreamDestroy(stream));

    CUDA_CALL(hipDeviceReset());

    return EXIT_SUCCESS;
}

void matrixInitialization(double *a, double *b, const size_t n) {
    assert_non_null(a);
    assert_non_null(b);

    for (size_t i = 0; i < n; i++) {
        for (size_t j = 0; j < n; j++) {
            a[i * n + j] = randomInRange(1, 1000);
            b[i * n + j] = randomInRange(1, 1000);
        }
    }
}
